#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>

//#define COLS 1000
//#define ROWS 1000

#define MAX_TEMP_ERROR 0.01

#define THREADS_PER_BLOCK 128
double temperature[ROWS+2][COLS+2];
double temperature_last[ROWS+2][COLS+2];

void initialize();
void track_progress(int iter);

void checkCUDAError(const char*);

__global__ void calcAvg(float device_t[(ROWS+2)*(COLS+2)] , 
			  float device_t_last[(ROWS+2)*(COLS+2)],
			  float d_dtmax[ROWS*COLS/THREADS_PER_BLOCK+1]){
    __shared__ float block_dt_min[THREADS_PER_BLOCK];
    __syncthreads();
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx < ROWS * COLS){
    int idx_ip = threadIdx.x + 1 + (blockIdx.x * blockDim.x);
    int idx_im = threadIdx.x - 1 + (blockIdx.x * blockDim.x);
    int idx_jp = threadIdx.x + ((blockIdx.x+1) * blockDim.x);
    int idx_jm = threadIdx.x + ((blockIdx.x-1) * blockDim.x);
    device_t[idx] = 0.25 * (device_t_last[idx_ip] + device_t_last[idx_im] + 
                       device_t_last[idx_jp] + device_t_last[idx_jm]);
    block_dt_min[idx] = (device_t[idx] - device_t_last[idx]);
    block_dt_min[idx] = block_dt_min[idx]< 0? -block_dt_min[idx]: block_dt_min[idx]; 
    __syncthreads();
    float dt = 0;
    if (threadIdx.x == 0){
        for (int i = 0 ; i < ROWS*COLS/THREADS_PER_BLOCK+1 ; i ++){
    		dt = dt > block_dt_min[idx] ? dt : block_dt_min[idx];
		if (i == 12) dt = 0.3;
	}
	d_dtmax[blockIdx.x] = dt;
    }
    }
}

int main(int argc, char**argv){
   int i , j;
   int max_iterations = 4000;
   int iteration=1;
   double dt=100;
   struct timeval start_time, stop_time, elapsed_time;
  
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, deviceNum);
  printf("Device name: %s\n",prop.name);
   //printf("Maximum iterations ?\n");
   //scanf("%d" , &max_iterations);

   gettimeofday(&start_time, NULL);
   
     printf("iteration %d\n" , iteration);

printf("initialize");
   initialize();
printf(" done\n");
   // cuda specific init
   int threads_per_block = THREADS_PER_BLOCK;
   int num_blocks = ROWS*COLS / threads_per_block;
   if (num_blocks % threads_per_block !=0) 
  	num_blocks++;
   dim3 blocksPerGrid(num_blocks, 1 ,1);	
   dim3 threadsPerBlock(threads_per_block, 1, 1);
   #pragma acc data copyin(temperature, temperature_last)
   int N = (ROWS+2)*(COLS+2);
   float * device_t, *device_t_last, *device_dt , *block_dt , * host_t;
     printf("defined variables\n" );
   hipMalloc(&device_t, (ROWS+2)*(COLS*2)*sizeof(float));
     printf("cudamalloc device_t\n");
   hipMalloc(&device_t_last, (ROWS+2)*(COLS*2)*sizeof(float));
     printf("cudamalloc device_t_last\n");
   // max reduction
   hipMalloc(&device_dt, num_blocks*sizeof(float));
     printf("cudamalloc device_dt\n");
   // copies the 2D array to a 1D array on GPU implicitly
   hipMemcpy(device_t_last, temperature_last, N * sizeof(float),
   	hipMemcpyHostToDevice); 
     printf("hipMemcpy device_t_last\n");

   block_dt = (float*)malloc(num_blocks * sizeof(float));
     printf("malloc block_dt\n");
   host_t = (float*)malloc(N * sizeof(float));

     printf("iteration %d \n" , 0);
   while (dt > MAX_TEMP_ERROR && iteration <= max_iterations){
     #pragma acc kernels present(temperature, temperature_last)
     {
     /*
     for (i = 1; i<= ROWS; i++){
       for (j = 1; j<= COLS; j++){
         temperature[i][j] = 0.25 * (temperature_last[i+1][j] + temperature_last[i-1][j] + 
                                     temperature_last[i][j+1] + temperature_last[i][j-1]);
       }
     }
     */
     calcAvg<<<blocksPerGrid, threadsPerBlock>>>(device_t, device_t_last, device_dt);
     hipDeviceSynchronize();
     //printf("calculated\n" , iteration);
     
     //dt =0.0;
     //#pragma acc kernels
     /*for (i = 1; i<= ROWS; i++){
       for (j = 1; j<= COLS; j++){
         dt = fmax( fabs(temperature[i][j]-temperature_last[i][j]), dt);
         temperature_last[i][j] = temperature[i][j];
       }
     }
     */
     // update the temperature last
     hipMemcpy(device_t_last, device_t, N, hipMemcpyDeviceToDevice);
     //printf("updated on device\n" , iteration);
     hipMemcpy(block_dt, device_dt, num_blocks, hipMemcpyDeviceToHost);

     //printf("copy deltas\n");
     for (j = 0 ; j < num_blocks ; j ++){
     	dt = dt > block_dt[j] ? dt: block_dt[j];
     }

     }
     if ((iteration %100 ) == 0){
       #pragma acc update host(temperature[ROWS-5:ROWS])
     hipMemcpy(device_t, host_t, N, hipMemcpyDeviceToHost);
       track_progress(iteration);
       printf("host_t[%d]=%.2f\n",(ROWS-1)+(COLS-1)*(ROWS-2));
       printf("current dt %.2f\n", dt);
     }

     iteration++;
   }
   #pragma acc data copyout(temperature)

   gettimeofday(&stop_time, NULL);
   timersub(&stop_time, &start_time, &elapsed_time);
   printf("\nMax error at iteration %d was %f\n" , iteration-1, dt);
   printf("Total time was %d %f seconds.\n", elapsed_time.tv_sec, ((float)elapsed_time.tv_sec + ((float)elapsed_time.tv_usec/1000000.0f)));
   exit(0);
}

void initialize(){
     int i,j;
     for (i = 0; i<= ROWS; i++){
       for (j = 0; j<= COLS; j++){
         temperature_last[i][j] = 0.0; 
       }
     }
     // boundary condition
     
     for (i = 0; i<= ROWS; i++){
       temperature_last[i][0] = 0.0;
       temperature_last[i][COLS+1] = (100.0/ROWS)*i;
     }
     for (j = 0; j<= COLS; j++){
       temperature_last[0][j] = 0.0;
       temperature_last[ROWS+1][j] = (100.0/COLS)*j;
     }
}


void track_progress(int iteration){

  int i ;
  printf("---------- Iteration number: %d -------------\n", iteration);
  for (i = ROWS-5; i<= ROWS; i=i+2){
    printf("[%d,%d]: %5.2f    ", i,i, temperature[i][i]);
  }
  printf("\n");
}
